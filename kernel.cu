#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "CUDA_memory.h"
#include "hip/hip_vector_types.h"
#include "printstring_helper.h"
#include "adaptive_mesh.cu"
#include "spacetime.h"

#include <stdio.h>

constexpr int max_lacunarity = 1 << 6;
constexpr int batch_size = 1 << 6;
constexpr int gpu_alloc_var = 1 << 20;
constexpr int domain_size = 16;

constexpr int gpu_alloc_dom = gpu_alloc_var / (domain_size * domain_size * domain_size);
constexpr int gpu_alloc_bds = gpu_alloc_dom * 6;

void sleep_forever()
{
    while (true)
        _sleep(1000000);
}

void check_regenerate_refine(BSSN_simulation& simulation, parent_node& tree, smart_gpu_cpu_buffer<octree_node_gpu>& temp_buffer, smart_gpu_cpu_buffer<octree_boundary_gpu>& boundaries) 
{
    if (!tree.is_dirty())
        return;

    int a = 0;
    tree.regenerate_boundaries();
    for (int l = tree.domains.size(), i = tree.new_child_index_start; i < l; i++)
        if (tree.domains[i] != nullptr)
            if (tree.domains[i]->newly_born())
            {
                temp_buffer.cpu_buffer_ptr[a] = octree_node_gpu(tree.domains[i]);
                a++;
            }
    temp_buffer.copy_to_gpu();

    AMR_refine_domain_batch(simulation.old_conformal_christoffel_trace, temp_buffer, tree, a);
    AMR_refine_domain_batch(simulation.old_conformal_metric, temp_buffer, tree, a);
    AMR_refine_domain_batch(simulation.old_extrinsic_curvature__lapse__conformal_factor, temp_buffer, tree, a);
    AMR_refine_domain_batch(simulation.old_shift_vector, temp_buffer, tree, a);
    AMR_refine_domain_batch(simulation.old_traceless_conformal_extrinsic_curvature, temp_buffer, tree, a);
}

void check_regenerate(BSSN_simulation& simulation, parent_node& tree, smart_gpu_cpu_buffer<octree_node_gpu>& domains, smart_gpu_cpu_buffer<octree_boundary_gpu>& boundaries)
{
    if (tree.removed_children < max_lacunarity)
        return;

    tree.regenerate_domains();
    AMR_yield_buffers(tree, domains, boundaries, true);

    AMR_copy_to(simulation.old_conformal_christoffel_trace, simulation.new_conformal_christoffel_trace, tree, domains);
    AMR_copy_to(simulation.old_conformal_metric, simulation.new_conformal_metric, tree, domains);
    AMR_copy_to(simulation.old_extrinsic_curvature__lapse__conformal_factor, simulation.new_extrinsic_curvature__lapse__conformal_factor, tree, domains);
    AMR_copy_to(simulation.old_shift_vector, simulation.new_shift_vector, tree, domains);
    AMR_copy_to(simulation.old_traceless_conformal_extrinsic_curvature, simulation.new_traceless_conformal_extrinsic_curvature, tree, domains);
    simulation.swap_old_new();
}

int main()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        sleep_forever();
        return 1;
    }
    
    parent_node parent = parent_node(make_uint3(domain_size));
    parent.add_child(parent.root, 0);
    parent.add_child(parent.root, 1);
    parent.add_child(parent.root, 3);
    parent.add_child(parent.root, 2);
    parent.add_child(parent.root, 5);
    parent.add_child(parent.root, 6);
    parent.add_child(parent.root, 4);
    parent.regenerate_boundaries();

    BSSN_simulation simulation = BSSN_simulation(gpu_alloc_var, (size_t)parent.stride * batch_size);
    smart_gpu_cpu_buffer<octree_node_gpu> temp = smart_gpu_cpu_buffer<octree_node_gpu>(gpu_alloc_dom);

    smart_gpu_cpu_buffer<octree_node_gpu> domains = smart_gpu_cpu_buffer<octree_node_gpu>(gpu_alloc_dom);
    smart_gpu_cpu_buffer<octree_boundary_gpu> boundaries = smart_gpu_cpu_buffer<octree_boundary_gpu>(gpu_alloc_bds);

    AMR_yield_buffers(parent, domains, boundaries, true);
    sleep_forever();

    temp.destroy();
    boundaries.destroy();
    domains.destroy();
    parent.destroy();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel program failed!");
        sleep_forever();
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        sleep_forever();
        return 1;
    }
    
    sleep_forever();
    return 0;
}
