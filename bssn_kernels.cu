#ifndef BSSN_KERNELS
#define BSSN_KERNELS

#include "hip/hip_runtime.h"
#include ""

#include "adaptive_mesh.cu"
#include "spacetime.h"


#endif
